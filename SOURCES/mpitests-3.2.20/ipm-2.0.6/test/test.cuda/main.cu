
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

#define REPEAT 10000

extern "C" __global__ void foo(float *a, int N)
{
  int i;
  
  for( i=0; i<REPEAT; i++ ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx<N) 
      a[idx] = a[idx] * a[idx];
  }
}


extern "C" __global__ void bar(float *a, int N)
{
  int i;
  
  for( i=0; i<REPEAT/100; i++ ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx<N) 
      a[idx] = a[idx] * a[idx];
  }
}

int main(int argc, char* argv[])
{
  const int N = 100000;   // Number of elements in arrays  
  float *a_h, *a_d;       // Pointer to host & device arrays
  float *b_h, *b_d;       // Pointer to host & device arrays
  size_t size = N * sizeof(float);
  

  a_h = (float *)malloc(size);        // Allocate array on host
  hipMalloc((void **) &a_d, size);   // Allocate array on device

  b_h = (float *)malloc(size);        // Allocate array on host
  hipMalloc((void **) &b_d, size);   // Allocate array on device
  
  // Initialize host array and copy it to CUDA device
  for (int i=0; i<N; i++) a_h[i] = 1+(float)1/i;
  

  // Do calculation on device:
  int block_size = 4;
  int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);

  hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);
  foo <<< n_blocks, block_size >>> (a_d, N);


  sleep(10);

  foo <<< n_blocks, block_size >>> (b_d, N);
  hipMemcpy(a_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);
  hipMemcpy(b_h, b_d, sizeof(float)*N, hipMemcpyDeviceToHost);

  // Print results
  //  for (int i=0; i<N; i++) printf("%d %f\n", i, a_h[i]);
  // Cleanup
  free(a_h); hipFree(a_d);
  free(b_h); hipFree(b_d);
  return 0;
}
